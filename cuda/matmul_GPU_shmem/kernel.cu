#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <stdio.h>

/* macro for index calculations */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* matrix size and thread dimensions */

#define SIZE 1024

/* define blocksize X and blocksize Y and blocksize K */

#define THREADS_PER_BLOCK_X 16 // Thread block size, x dimension
#define THREADS_PER_BLOCK_Y 16 // Thread block size, y dimension
#define BK 16 // square block of K size

__global__ void GPU_shmem(const int m, double const * const a, double const * const b, double *c )
{

/* setup some constanst for later use */

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int iby = blockIdx.y * THREADS_PER_BLOCK_Y;
	const int ibx = blockIdx.x * THREADS_PER_BLOCK_X;

/* shared memory arrays for A and B */

	__shared__ double as[ THREADS_PER_BLOCK_X ][ BK + 1 ];
	__shared__ double bs[ BK ][ THREADS_PER_BLOCK_Y + 1 ];
	
/* space for C to be held in registers */

	double c_tmp = 0.0 ;

	/* calculate my initial offset into A and B */

	int aoff = INDX( ibx + tx, ty, m );
	int boff = INDX( tx, iby + ty, m );

	/* main loop over blocks of K */

	for( int Kblock = 0; Kblock < m; Kblock+=BK )
	{

		/* read block of A into shared memory */

			as[ tx ][ ty ] = a[ aoff ];

		/* read block of B into shared memory */

			bs[ tx ][ ty ] = b[ boff ];
		

		__syncthreads();

		/* increment A and B offsets  for next round of data reads */

		boff += BK;
		aoff += m * BK;

		/* triply nested loop to perform the matmult on the blocks */

#pragma unroll
		for( int k = 0 ; k < BK ; k++ )
		{
			c_tmp += as[ tx ][ k ] * bs[ k ][ ty ];
		}
		__syncthreads();

	} /* end for Kblock */

	/* set C to its proper index int the C matrix */

	int coff = INDX( ibx + tx, iby + ty, m );

	/* write results to the C matrix */

	c[ coff ] = c_tmp;
 
} /* end GPU_shmem */


int main( int argc, char *argv[] )
{

    const int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

    double *h_a, *h_b, *h_c, *h_c1;
    double *d_a, *d_b, *d_c;
 
    size_t numbytes = (size_t ) size * (size_t ) size * sizeof( double );

    h_a = (double *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_b = (double *) malloc( numbytes );
    if( h_b == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_c = (double *) malloc( numbytes );
    if( h_c == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	h_c1 = (double *) malloc( numbytes );
    if( h_c1 == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	/* zero out the host memory for C matrices */

    memset( h_c, 0, numbytes );
    memset( h_c1, 0, numbytes );

    fprintf( stdout, "Total memory required is %lf MB\n", 
       3.0 * (double) numbytes / 1000000.0 );

	/* initialize the A and B matrices */

    for( int i = 0; i < size * size; i++ )
    {
      h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
      h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    }

	/* allocate a, b, c in gpu memory */

    hipMalloc( (void **)&d_a, numbytes );
    hipMalloc( (void **)&d_b, numbytes );
    hipMalloc( (void **)&d_c, numbytes );
	
	/* copy a and b to device */

	hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice );
    hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice );

    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate( &handle );

    double alpha = 1.0;
    double beta  = 0.0;

	/* start timers */

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

	/* call CUBLAS dgemm */

hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha, 
                 d_a, size,
                 d_b, size,
                 &beta,
                 d_c, size );

	/* stop timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );

	/* print GPU CUBLAS timing information */

    fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
    
	/* copy C from device to host for error checking */

    hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost );

	/* reset C on device to zero */

	hipMemset( d_c, 0, numbytes );

	/* setup grid and block sizes */

     dim3 blocksize( THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1 );
     dim3 gridsize( size / blocksize.x, size / blocksize.y, 1 );

	/* start timers */

	hipEventRecord( start, 0 );

	/* call GPU_naive */

	GPU_shmem<<< gridsize, blocksize >>> ( size, d_a, d_b, d_c );

	/* stop timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );

	/* print data for GPU naive */

    fprintf(stdout, "Total time GPU SHMEM is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
	/* copy C back to host */
	
	hipMemcpy( h_c1, d_c, numbytes, hipMemcpyDeviceToHost );

    hipblasDestroy( handle );
    hipEventDestroy( start );
    hipEventDestroy( stop );

	/* check CUBLAS versus GPU NAIVE numerical results */

	double temp = 0.0;

	for( int i = 0; i < size * size; i++ )
	{
		temp += ( h_c[i] - h_c1[i] ) * ( h_c[i] - h_c1[i] );
	} /* end for */

	printf("error is %f\n",temp);
	if( temp > 10 ) printf("Error value is suspiciously high!\n");

	/* cleanup */

    hipFree( d_a );
    hipFree( d_b );
	hipFree( d_c );

    free( h_a );
    free( h_b );
    free( h_c );
    free( h_c1 );

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
