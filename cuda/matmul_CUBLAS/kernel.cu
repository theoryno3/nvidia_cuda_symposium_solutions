#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <stdio.h>

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

#define SIZE 1024

void host_dgemm( int m, int n, int k, double *a, double *b, double *c )
{

  for( int j = 0; j < n; j++ )
  {
	for( int i = 0; i < m; i++ )
	{
		for( int koff = 0; koff < k; koff++ )
		{
			c[INDX(i, j, m)] += a[INDX( i, koff, m )] * b[INDX( koff, j, n )];
		} /* end for i */
	  } /* end jb */
  } /* end for j */

} /* end host_dgemm */

int main( int argc, char *argv[] )
{

    const int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

    double *h_a, *h_b, *h_c, *h_cdef;
    double *d_a, *d_b, *d_c;
 
    size_t numbytes = size * size * sizeof( double );

    h_a = (double *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_b = (double *) malloc( numbytes );
    if( h_b == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_c = (double *) malloc( numbytes );
    if( h_c == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_cdef = (double *) malloc( numbytes );
    if( h_cdef == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    memset( h_c, 0, numbytes );
    memset( h_cdef, 0, numbytes );

    fprintf( stdout, "Total memory required is %lf MB\n", 
       3.0 * (double) numbytes / 1000000.0 );

    for( int i = 0; i < size * size; i++ )
    {
      h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
      h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    }

    hipMalloc( (void **)&d_a, numbytes );
    hipMalloc( (void **)&d_b, numbytes );
    hipMalloc( (void **)&d_c, numbytes );

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );


    hipEventRecord( start, 0 );

    host_dgemm( size, size, size, h_a, h_b, h_cdef );

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );

    fprintf(stdout, "Total time CPU is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );


    hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice );
    hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice );

    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate( &handle );

    double alpha = 1.0;
    double beta  = 0.0;

    hipEventRecord( start, 0 );

    hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha, 
                 d_a, size,
                 d_b, size,
                 &beta,
                 d_c, size );


    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime, start, stop );

    fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
    hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost );

    hipblasDestroy( handle );
    hipEventDestroy( start );
    hipEventDestroy( stop );

	double temp = 0.0;

	for( int i = 0; i < size * size; i++ )
	{
		temp += ( h_c[i] - h_cdef[i] ) * ( h_c[i] - h_cdef[i] );
	} /* end for */

	printf("error is %f\n",temp);
	if( temp > 10 ) printf("Error value is suspiciously high!\n");

    hipFree( d_a );
    hipFree( d_b );
    hipFree( d_c );

    free( h_a );
    free( h_b );
    free( h_c );
    free( h_cdef );

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
