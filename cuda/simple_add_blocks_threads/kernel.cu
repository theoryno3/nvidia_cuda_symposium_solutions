#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
    int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */

	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */

	add<<< N / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, d_b, d_c );

	/* copy result back to host */

	hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost );


	printf( "c[0] = %d\n",0,c[0] );
	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
