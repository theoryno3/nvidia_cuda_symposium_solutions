#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    *c = *a + *b;
}

int main()
{
    int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof( int );

	/* allocate space for device copies of a, b, c */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* setup initial values */

	a = 2;
	b = 7;
	c = -99;

	/* copy inputs to device */

	hipMemcpy( d_a, &a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_b, &b, size, hipMemcpyHostToDevice );

	/* launch the kernel on the GPU */

	add<<< 1, 1 >>>( d_a, d_b, d_c );

	/* copy result back to host */

	hipMemcpy( &c, d_c, size, hipMemcpyDeviceToHost );

	printf("value of c after kernel is %d\n",c);

	/* clean up */

	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
