#include "hip/hip_runtime.h"

#include <stdio.h>

#define N 4000000
#define RADIUS 5
#define THREADS_PER_BLOCK 512

__global__ void stencil_1d(int n, double *in, double *out)
{
	/* calculate global index in the array */
	int gindex = blockIdx.x * blockDim.x + threadIdx.x;
	
	/* return if my global index is larger than the array size */
	if( gindex >= n ) return;

	/* code to handle the boundary conditions */
	if( gindex < RADIUS || gindex >= (n - RADIUS) ) 
	{
		out[gindex] = (double) gindex * ( (double)RADIUS*2 + 1) ;
		return;
	} /* end if */


	double result = 0.0;
	
	for( int i = gindex-(RADIUS); i <= gindex+(RADIUS); i++ ) 
	{
		result += in[i];
	}

	out[gindex] = result;
	return;
}

int main()
{
    double *in, *out;
	double *d_in, *d_out;
	int size = N * sizeof( double );

	/* allocate space for device copies of in, out */

	hipMalloc( (void **) &d_in, size );
	hipMalloc( (void **) &d_out, size );

	/* allocate space for host copies of in, out and setup input values */

	in = (double *)malloc( size );
	out = (double *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		in[i] = (double) i;
		out[i] = 0;
	}

	/* copy inputs to device */

	hipMemcpy( d_in, in, size, hipMemcpyHostToDevice );
	hipMemset( d_out, 0, size );

	/* calculate block and grid sizes */

	dim3 blocksize( THREADS_PER_BLOCK, 1, 1);
	dim3 gridsize( (N / blocksize.x) + 1, 1, 1);

	/* start the timers */

	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );
	hipEventRecord( start, 0 );

	/* launch the kernel on the GPU */

	stencil_1d<<< gridsize, blocksize >>>( N, d_in, d_out );

	/* stop the timers */

	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );

	printf("Total time for %d elements was %f ms\n", N, elapsedTime );

	/* copy result back to host */

	hipMemcpy( out, d_out, size, hipMemcpyDeviceToHost );

	for( int i = 0; i < N; i++ )
	{
		if( in[i]*( (double)RADIUS*2+1 ) != out[i] ) printf("error in element %d in = %f out %f\n",i,in[i],out[i] );
	} /* end for */

	/* clean up */

	free(in);
	free(out);
	hipFree( d_in );
	hipFree( d_out );
	
	return 0;
} /* end main */
