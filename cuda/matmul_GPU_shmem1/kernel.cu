#include "hip/hip_runtime.h"
#include "hipblas.h"

#include <stdio.h>

/* macro for index calculations */

#define INDX( row, col, ld ) ( ( (col) * (ld) ) + (row) )

/* matrix size and thread dimensions */

#define SIZE 1024

/* setup various hard-coded parameters for this kernel */

#define TBX 64 // Size of C this CTA is responsible for, x dimension
#define TBY 64 // Size of C this CTA is responsible for, y dimension
#define THREADS_PER_BLOCK_X 16 // Thread block size, x dimension
#define THREADS_PER_BLOCK_Y 16 // Thread block size, y dimension
#define BK 16 // square block of K size
#define NX 4  // = TBX/THREADS_PER_BLOCK_X == number of iterations to do TBX work with THREADS_PER_BLOCK_X blocks
#define NY 4  // = TBY/THREADS_PER_BLOCK_Y == number of iterations to do TBY work with THREADS_PER_BLOCK_Y blocks

__global__ void GPU_shmem1(const int m, double const * const a, double const * const b, double *c )
{

/* setup some constants for later use */

	const int tx = threadIdx.x;
	const int ty = threadIdx.y;
	const int iby = blockIdx.y * TBY;
	const int ibx = blockIdx.x * TBX;

/* shared memory arrays for A and B */

	__shared__ double as[ TBX ][ BK+1 ];
	__shared__ double bs[ BK ][ TBY+1 ];
	
/* space for C to be held in registers */

	double c_tmp[ NX ][ NY ] ;

	/* zero the temp C array */

#pragma unroll
	for ( int i = 0 ; i < NX ; i++) { 
		for ( int j = 0 ; j < NY ; j++) {
			c_tmp[i][j] = 0.0;
		}
	}

	/* calculate my initial offset into A and B */

	int aoff = INDX( ibx + tx, ty, m );
	int boff = INDX( tx, iby + ty, m );

	/* main loop over blocks of K */

	for( int Kblock = 0; Kblock < m; Kblock+=BK )
	{

		/* read block of A into shared memory */

#pragma unroll
		for ( int i = 0; i < NX ; i ++ ) 
		{
			as[ tx + i * THREADS_PER_BLOCK_X ][ ty ] = a[ (aoff + i*THREADS_PER_BLOCK_X) ];
		}

		/* read block of B into shared memory */

#pragma unroll
		for ( int i = 0; i < NY ; i ++ ) 
		{
			bs[ tx ][ ty + THREADS_PER_BLOCK_Y * i ] = b[ (boff + m*i*THREADS_PER_BLOCK_Y) ];
		}

		__syncthreads();

		/* increment A and B offsets  for next round of data reads */

		boff += BK;
		aoff += m * BK;

		/* triply nested loop to perform the matmult on the blocks */

#pragma unroll
		for( int k = 0 ; k < BK ; k++ )
		{
#pragma unroll
			for (int j = 0 ; j < NY ; j++ )
			{
#pragma unroll
				for (int i = 0 ; i < NX ; i++ )
				{
					c_tmp[ i ][ j ] += as[ tx + THREADS_PER_BLOCK_X*i ][ k ] * bs[ k ][ ty + j*THREADS_PER_BLOCK_Y ];
				}
			}
		}
		__syncthreads();

	} /* end for Kblock */

	/* set coff to its proper index int the C matrix */

	int coff = INDX( ibx + tx, iby + ty, m );
  
	/* write results to the C matrix */

#pragma unroll
	for ( int j = 0 ; j < NY ; j++ ) 
	{
#pragma unroll
		for ( int i = 0 ; i < NX ; i++ )
		{      
			c[ coff + INDX( THREADS_PER_BLOCK_X * i, THREADS_PER_BLOCK_Y * j, m )] = c_tmp[i][j];
		}
	}
 
} /* end GPU_shmem1 */

int main( int argc, char *argv[] )
{

    const int size = SIZE;

    fprintf(stdout, "Matrix size is %d\n",size);

    double *h_a, *h_b, *h_c, *h_c1;
    double *d_a, *d_b, *d_c;
 
    size_t numbytes = (size_t ) size * (size_t ) size * sizeof( double );

    h_a = (double *) malloc( numbytes );
    if( h_a == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_b = (double *) malloc( numbytes );
    if( h_b == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

    h_c = (double *) malloc( numbytes );
    if( h_c == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	h_c1 = (double *) malloc( numbytes );
    if( h_c1 == NULL )
    {
      fprintf(stderr,"Error in host malloc\n");
      return 911;
    }

	/* zero out the host memory for C matrices */

    memset( h_c, 0, numbytes );
    memset( h_c1, 0, numbytes );

    fprintf( stdout, "Total memory required is %lf MB\n", 
       3.0 * (double) numbytes / 1000000.0 );

	/* initialize the A and B matrices */

    for( int i = 0; i < size * size; i++ )
    {
      h_a[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
      h_b[i] = double( rand() ) / ( double(RAND_MAX) + 1.0 );
    }

	/* allocate a, b, c in gpu memory */

    hipMalloc( (void **)&d_a, numbytes );
    hipMalloc( (void **)&d_b, numbytes );
    hipMalloc( (void **)&d_c, numbytes );
	
	/* copy a and b to device */

	hipMemcpy( d_a, h_a, numbytes, hipMemcpyHostToDevice );
    hipMemcpy( d_b, h_b, numbytes, hipMemcpyHostToDevice );

    hipblasHandle_t handle;
    hipblasStatus_t stat = hipblasCreate( &handle );

    double alpha = 1.0;
    double beta  = 0.0;

	/* start timers */

    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop );
    hipEventRecord( start, 0 );

	/* call CUBLAS dgemm */

hipblasDgemm( handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha, 
                 d_a, size,
                 d_b, size,
                 &beta,
                 d_c, size );

	/* stop timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
    float elapsedTime;
    hipEventElapsedTime( &elapsedTime, start, stop );

	/* print GPU CUBLAS timing information */

    fprintf(stdout, "Total time GPU CUBLAS is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
    
	/* copy C from device to host for error checking */

    hipMemcpy( h_c, d_c, numbytes, hipMemcpyDeviceToHost );

	/* reset C on device to zero */

	hipMemset( d_c, 0, numbytes );

	/* setup grid and block sizes */

	dim3 blocksize( THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y, 1 );
	dim3 gridsize( size / ( TBX ), size / ( TBY ), 1 );	

	/* call GPU_naive */

	printf("block.X %d block.Y %d\n",gridsize.x, gridsize.y );
	printf("threads.x %d threads.y %d\n",blocksize.x, blocksize.y );
    
/* start timers */

	hipEventRecord( start, 0 );

/* call the kernel */

	GPU_shmem1<<< gridsize, blocksize >>> ( size, d_a, d_b, d_c );

	/* stop timers */

    hipEventRecord( stop, 0 );
    hipEventSynchronize( stop );
	elapsedTime = 0.0f;
    hipEventElapsedTime( &elapsedTime, start, stop );

	/* print data for GPU naive */

    fprintf(stdout, "Total time GPU SHMEM is %f sec\n", elapsedTime / 1000.0f );
    fprintf(stdout, "Performance is %f GFlop/s\n", 
      2.0 * (double) size * (double) size * (double) size / 
      ( (double) elapsedTime / 1000.0 ) * 1.e-9 );
                  
	/* copy C back to host */
	
	hipMemcpy( h_c1, d_c, numbytes, hipMemcpyDeviceToHost );

    hipblasDestroy( handle );
    hipEventDestroy( start );
    hipEventDestroy( stop );

	/* check CUBLAS versus GPU NAIVE numerical results */

	double temp = 0.0;

	for( int i = 0; i < size * size; i++ )
	{
		temp += ( h_c[i] - h_c1[i] ) * ( h_c[i] - h_c1[i] );
      //  printf("i %d h_c %f h_c1 %f\n",i,h_c[i],h_c1[i] );
	} /* end for */

	printf("error is %f\n",temp);
	if( temp > 10 ) printf("Error value is suspiciously high!\n");

	/* cleanup */

    hipFree( d_a );
    hipFree( d_b );
	hipFree( d_c );

    free( h_a );
    free( h_b );
    free( h_c );
    free( h_c1 );

    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}
